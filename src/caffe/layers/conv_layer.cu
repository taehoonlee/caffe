#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void ConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data;
    Dtype* top_data;
    if (this->usingdata2) {
      bottom_data = bottom[i]->gpu_data2();
      top_data = top[i]->mutable_gpu_data2();
    } else {
      bottom_data = bottom[i]->gpu_data();
      top_data = top[i]->mutable_gpu_data();
    }
    for (int n = 0; n < this->num_; ++n) {
      this->forward_gpu_gemm(bottom_data + bottom[i]->offset(n), weight,
          top_data + top[i]->offset(n));
      if (this->bias_term_) {
        const Dtype* bias = this->blobs_[1]->gpu_data();
        this->forward_gpu_bias(top_data + top[i]->offset(n), bias);
      }
    }
  }
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = this->blobs_[0]->gpu_data();
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff;
    Dtype* bottom_diff;
    if (this->adversarial) {
      top_diff = top[i]->gpu_diff2();
      bottom_diff = bottom[i]->mutable_gpu_diff2();
    } else {
      top_diff = top[i]->gpu_diff();
      bottom_diff = bottom[i]->mutable_gpu_diff();
    }
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1] && !this->adversarial) {
      if (this->manifold) {
        for (int n = 0; n < this->num_; ++n) {
          this->backward_gpu_bias(this->blobs_[1]->mutable_gpu_diff2(), top[i]->gpu_diff2() + top[i]->offset(n));
          this->backward_gpu_bias(this->blobs_[1]->mutable_gpu_diff3(), top[i]->gpu_diff3() + top[i]->offset(n));
        }
      } else {
        for (int n = 0; n < this->num_; ++n) {
          this->backward_gpu_bias(this->blobs_[1]->mutable_gpu_diff(), top_diff + top[i]->offset(n));
        }
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      for (int n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0] && !this->adversarial) {
          if (this->manifold) {
            this->weight_gpu_gemm(bottom[i]->gpu_data() + bottom[i]->offset(n),
                top[i]->gpu_diff2() + top[i]->offset(n), this->blobs_[0]->mutable_gpu_diff2());
            this->weight_gpu_gemm(bottom[i]->gpu_data2() + bottom[i]->offset(n),
                top[i]->gpu_diff3() + top[i]->offset(n), this->blobs_[0]->mutable_gpu_diff3());
          } else {
            this->weight_gpu_gemm(bottom[i]->gpu_data() + bottom[i]->offset(n),
                top_diff + top[i]->offset(n), this->blobs_[0]->mutable_gpu_diff());
          }
        }
        // gradient w.r.t. bottom data, if necessary.
        if (propagate_down[i]) {
          if (this->manifold) {
            this->backward_gpu_gemm(top[i]->gpu_diff2() + top[i]->offset(n), weight,
                bottom[i]->mutable_gpu_diff2() + bottom[i]->offset(n));
            this->backward_gpu_gemm(top[i]->gpu_diff3() + top[i]->offset(n), weight,
                bottom[i]->mutable_gpu_diff3() + bottom[i]->offset(n));
          } else {
            this->backward_gpu_gemm(top_diff + top[i]->offset(n), weight,
                bottom_diff + bottom[i]->offset(n));
          }
        } else if (this->adversarial) { // TAEHOON : n is the sample index
          this->backward_gpu_gemm(top_diff + top[i]->offset(n), weight,
              bottom_diff + bottom[i]->offset(n));
          Dtype sumsq;
          caffe_gpu_dot(784, bottom_diff + bottom[i]->offset(n), bottom_diff + bottom[i]->offset(n), &sumsq);
          if ( sumsq < 1e-9 )
            sumsq = (Dtype) 0.0;
          else
            sumsq = (Dtype) 1.0 / sqrt(sumsq);
          caffe_gpu_scal(784, sumsq, bottom_diff + bottom[i]->offset(n));
        }
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionLayer);

}  // namespace caffe
